#include "hip/hip_runtime.h"

/*
	2018.01.20
	Created by AcrylicShrimp.
*/

#include "TinNet_GPU_Backend.h"

#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"
#include "hip/hip_runtime.h"
#include "device_atomic_functions.h"
#include "hip/device_functions.h"
#include ""
#include "math_functions.h"

#if (_DEBUG)
#include <cstdio>
#endif

#pragma region Kernel

__global__ void kernel_mergeBatch(unsigned int nBatchSize, unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;

	if (nUnifiedIndex >= nSize)
		return;

	for (unsigned int nBatch = 1; nBatch < nBatchSize; ++nBatch)
		pOutput[nUnifiedIndex] += pInput[nSize * nBatch + nUnifiedIndex];
}

__global__ void kernel_update(unsigned int nParamSize, const float *pParamDelta, float *pParam)
{
	unsigned int nUnifiedOut = blockDim.x * blockIdx.x + threadIdx.x;

	if (nUnifiedOut >= nParamSize)
		return;

	pParam[nUnifiedOut] += pParamDelta[nUnifiedOut];
}

__global__ void kernel_updateFactor(unsigned int nParamSize, const float *pParamDelta, float *pParam, float nFactor)
{
	unsigned int nUnifiedOut = blockDim.x * blockIdx.x + threadIdx.x;

	if (nUnifiedOut >= nParamSize)
		return;

	pParam[nUnifiedOut] += nFactor * pParamDelta[nUnifiedOut];
}



__global__ void kernel_MSE_GPU_derivative(unsigned int nSize, unsigned int nOutputSize, const float *pOutput, const float *pDesiredOutput, float *pResult)
{
	unsigned int nUnifiedOut = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedOut
	};

	nUnifiedOut = vMax[nUnifiedOut < nSize];
	pResult[nUnifiedOut] = 2.f / nOutputSize * (pOutput[nUnifiedOut] - pDesiredOutput[nUnifiedOut]);
}

__global__ void kernel_CE_GPU_derivative(unsigned int nSize, unsigned int nOutputSize, const float *pOutput, const float *pDesiredOutput, float *pResult)
{
	unsigned int nUnifiedOut = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedOut
	};

	nUnifiedOut = vMax[nUnifiedOut < nSize];
	pResult[nUnifiedOut] = (pOutput[nUnifiedOut] - pDesiredOutput[nUnifiedOut]) / (pOutput[nUnifiedOut] - pOutput[nUnifiedOut] * pOutput[nUnifiedOut] + 1e-4f);
}

__global__ void kernel_MulticlassCE_GPU_derivative(unsigned int nSize, unsigned int nOutputSize, const float *pOutput, const float *pDesiredOutput, float *pResult)
{
	unsigned int nUnifiedOut = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedOut
	};

	nUnifiedOut = vMax[nUnifiedOut < nSize];
	pResult[nUnifiedOut] = -pDesiredOutput[nUnifiedOut] / (pOutput[nUnifiedOut] + 1e-4f);
}



__global__ void kernel_ConvLayer_GPU_forward(
	unsigned int nWidth,
	unsigned int nHeight,
	unsigned int nChannel,
	unsigned int nFilterWidth,
	unsigned int nFilterHeight,
	unsigned int nStrideHorizontal,
	unsigned int nStrideVertical,
	unsigned int nZeroPaddingHorizontalNegative,
	unsigned int nZeroPaddingVerticalNegative,
	const float *pBias, const float *pWeight, const float *pInput, float *pOutput)
{
	float nValue = pBias[blockIdx.x];
	
	for (unsigned int nFilterY = 0; nFilterY < nFilterHeight; ++nFilterY)
	{
		const unsigned int nY = threadIdx.y * nStrideVertical + nFilterY;

		if (nY < nZeroPaddingVerticalNegative)
			continue;

		if (nY >= nZeroPaddingVerticalNegative + nHeight)
			continue;

		const unsigned int nInputY = nY - nZeroPaddingVerticalNegative;

		for (unsigned int nFilterX = 0; nFilterX < nFilterWidth; ++nFilterX)
		{
			const unsigned int nX = threadIdx.x * nStrideHorizontal + nFilterX;

			if (nX < nZeroPaddingHorizontalNegative)
				continue;

			if (nX >= nZeroPaddingHorizontalNegative + nWidth)
				continue;

			const unsigned int nInputX = nX - nZeroPaddingHorizontalNegative;

			for (unsigned int nChannelIndex = 0; nChannelIndex < nChannel; ++nChannelIndex)
				nValue +=
					pInput[nChannelIndex * nWidth * nHeight + nInputY * nWidth + nInputX] *
				pWeight[blockIdx.x * nChannel * nFilterWidth * nFilterHeight + nChannelIndex * nFilterWidth * nFilterHeight + nFilterY * nFilterWidth + nFilterX];
		}
	}

	pOutput[blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x] = nValue;
}

__global__ void kernel_ConvLayer_GPU_forwardBatch(
	unsigned int nWidth,
	unsigned int nHeight,
	unsigned int nChannel,
	unsigned int nFilterWidth,
	unsigned int nFilterHeight,
	unsigned int nStrideHorizontal,
	unsigned int nStrideVertical,
	unsigned int nZeroPaddingHorizontalNegative,
	unsigned int nZeroPaddingVerticalNegative,
	const float *pBias, const float *pWeight, const float *pInput, float *pOutput)
{
	float nValue = pBias[blockIdx.x];

	for (unsigned int nFilterY = 0; nFilterY < nFilterHeight; ++nFilterY)
	{
		const unsigned int nY = threadIdx.y * nStrideVertical + nFilterY;

		if (nY < nZeroPaddingVerticalNegative)
			continue;

		if (nY >= nZeroPaddingVerticalNegative + nHeight)
			continue;

		const unsigned int nInputY = nY - nZeroPaddingVerticalNegative;

		for (unsigned int nFilterX = 0; nFilterX < nFilterWidth; ++nFilterX)
		{
			const unsigned int nX = threadIdx.x * nStrideHorizontal + nFilterX;

			if (nX < nZeroPaddingHorizontalNegative)
				continue;

			if (nX >= nZeroPaddingHorizontalNegative + nWidth)
				continue;

			const unsigned int nInputX = nX - nZeroPaddingHorizontalNegative;

			nValue = .0f;

			for (unsigned int nChannelIndex = 0; nChannelIndex < nChannel; ++nChannelIndex)
				nValue += pInput[blockIdx.y * nChannel * nWidth * nHeight + nChannelIndex * nWidth * nHeight + nInputY * nWidth + nInputX] * pWeight[blockIdx.x * nChannel * nFilterWidth * nFilterHeight + nChannelIndex * nFilterWidth * nFilterHeight + nFilterY * nFilterWidth + nFilterX];
		}
	}

	pOutput[blockIdx.y * gridDim.x * blockDim.x * blockDim.y + blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x] = nValue;
}

__global__ void kernel_ConvLayer_GPU_backwardBatch(
	unsigned int nWidth,
	unsigned int nHeight,
	unsigned int nChannel,
	unsigned int nFilterWidth,
	unsigned int nFilterHeight,
	unsigned int nStrideHorizontal,
	unsigned int nStrideVertical,
	unsigned int nZeroPaddingHorizontalNegative,
	unsigned int nZeroPaddingVerticalNegative,
	const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput, float *pBiasDelta, float *pWeightDelta, const float *pWeight)
{
	float nBias = .0f;

	for(unsigned int nOutputIndex = 0, nOutputSize = blockDim.x * blockDim.y; nOutputIndex < nOutputSize; ++nOutputIndex)
		nBias += pBackwardInput[blockIdx.y * gridDim.x * nOutputSize + blockIdx.x * nOutputSize + nOutputIndex];

	pBiasDelta[blockIdx.y * gridDim.x + blockIdx.x] = nBias;

	for (unsigned int nFilterY = 0; nFilterY < nFilterHeight; ++nFilterY)
	{
		const unsigned int nY = threadIdx.y * nStrideVertical + nFilterY;

		if (nY < nZeroPaddingVerticalNegative)
			continue;

		if (nY >= nZeroPaddingVerticalNegative + nHeight)
			continue;

		const unsigned int nInputY = nY - nZeroPaddingVerticalNegative;

		for (unsigned int nFilterX = 0; nFilterX < nFilterWidth; ++nFilterX)
		{
			const unsigned int nX = threadIdx.x * nStrideHorizontal + nFilterX;

			if (nX < nZeroPaddingHorizontalNegative)
				continue;

			if (nX >= nZeroPaddingHorizontalNegative + nWidth)
				continue;

			const unsigned int nInputX = nX - nZeroPaddingHorizontalNegative;

			for (unsigned int nChannelIndex = 0; nChannelIndex < nChannel; ++nChannelIndex)
			{
				atomicAdd(&pWeightDelta[blockIdx.y * gridDim.x * nChannel * nFilterWidth * nFilterHeight + blockIdx.x * nChannel * nFilterWidth * nFilterHeight + nChannelIndex * nFilterWidth * nFilterHeight + nFilterY * nFilterWidth + nFilterX],
					pBackwardInput[blockIdx.y * gridDim.x * blockDim.x * blockDim.y + blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.y * threadIdx.x] *
					pForwardInput[blockIdx.y * nChannel * nWidth * nHeight + nChannelIndex * nWidth * nHeight + nInputY * nWidth + nInputX]);

				atomicAdd(&pBackwardOutput[blockIdx.y * nChannel * nWidth * nHeight + nChannelIndex * nWidth * nHeight + nInputY * nWidth + nInputX],
					pBackwardInput[blockIdx.y * gridDim.x * blockDim.x * blockDim.y + blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.y * threadIdx.x] *
					pWeight[blockIdx.x * nChannel * nFilterWidth * nFilterHeight + nChannelIndex * nFilterWidth * nFilterHeight + (nFilterHeight - nFilterY - 1) * nFilterWidth + (nFilterWidth - nFilterX - 1)]);
			}
		}
	}
}

//__global__ void kernel_ConvLayer_GPU_mergeWeightDelta(
//	unsigned int nChannel,
//	unsigned int nOutputWidth,
//	unsigned int nOutputHeight,
//	float *pWeightDelta)
//{
//	for (unsigned int nOutputIndex = 0, nOutputSize = nOutputWidth * nOutputHeight; nOutputIndex < nOutputSize; ++nOutputIndex)
//		for (unsigned int nChannelIndex = 0; nChannelIndex < nChannel; ++nChannelIndex)
//			pWeightDelta[blockIdx.y * (gridDim.x + 1) * nOutputSize * nChannel * blockDim.x * blockDim.y + nOutputIndex * nChannel * blockDim.x * blockDim.y + nChannelIndex * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x] = .0f;
//
//	for (unsigned int nOutputY = 0; nOutputY < nOutputHeight; ++nOutputY)
//	for (unsigned int nOutputX = 0; nOutputX < nOutputWidth; ++nOutputX)
//		pWeightDelta[blockIdx.y * (gridDim.x + 1) * nOutputWidth * nOutputHeight * nChannel * blockDim.x * blockDim.y] +=
//}

__global__ void kernel_ConvLayer_GPU_Merge(
	unsigned int nWidth,
	unsigned int nHeight,
	unsigned int nChannel,
	unsigned int nFilterWidth,
	unsigned int nFilterHeight,
	unsigned int nStrideHorizontal,
	unsigned int nStrideVertical,
	unsigned int nZeroPaddingHorizontalNegative,
	unsigned int nZeroPaddingVerticalNegative,
	float *pBackwardInput, float *pWeightDelta)
{

}



__global__ void kernel_FullLayer_GPU_forward(unsigned int nInputSize, unsigned int nOutputSize, const float *pBias, const float *pWeight, const float *pInput, float *pOutput)
{
	extern __shared__ float pCopiedInput[];

	for (unsigned int nIn = threadIdx.x; nIn < nInputSize; nIn += blockDim.x)
		pCopiedInput[nIn] = pInput[nIn];

	__syncthreads();

	unsigned int nUnifiedOut = blockDim.x * blockIdx.x + threadIdx.x;

	if (nUnifiedOut >= nOutputSize)
		return;

	float nValue = pBias[nUnifiedOut];

	unsigned int nWeightIndex = nUnifiedOut * nInputSize;

	for (unsigned int nIn = 0; nIn < nInputSize; ++nIn)
		nValue += pCopiedInput[nIn] * pWeight[nWeightIndex + nIn];

	pOutput[nUnifiedOut] = nValue;
}

__global__ void kernel_FullLayer_GPU_forwardBatch(unsigned int nInputSize, unsigned int nOutputSize, const float *pBias, const float *pWeight, const float *pInput, float *pOutput)
{
	extern __shared__ float pCopiedInput[];
	
	for (unsigned int nIn = threadIdx.x; nIn < nInputSize; nIn += blockDim.x)
		pCopiedInput[nIn] = pInput[nInputSize * blockIdx.x + nIn];
	
	__syncthreads();
	
	unsigned int nUnifiedOut = blockDim.x * blockIdx.y + threadIdx.x;
	
	if (nUnifiedOut >= nOutputSize)
		return;
	
	float nValue = pBias[nUnifiedOut];
	
	for (unsigned int nIn = 0; nIn < nInputSize; ++nIn)
		nValue += pCopiedInput[nIn] * pWeight[nUnifiedOut * nInputSize + nIn];
		//nValue += pInput[blockIdx.x * nInputSize + nIn] * pWeight[nUnifiedOut * nInputSize + nIn];
	
	pOutput[nOutputSize * blockIdx.x + nUnifiedOut] = nValue;
}

__global__ void kernel_FullLayer_GPU_backwardBatch(unsigned int nInputSize, unsigned int nOutputSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput, float *pBiasDelta, float *pWeightDelta, const float *pWeight)
{
	extern __shared__ float pCopiedInput[];

	for (unsigned int nOut = threadIdx.x; nOut < nOutputSize; nOut += blockDim.x)
		pCopiedInput[nOut] = pBackwardInput[nOutputSize * blockIdx.x + nOut];
	
	for (unsigned int nIn = threadIdx.x; nIn < nInputSize; nIn += blockDim.x)
		pCopiedInput[nOutputSize + nIn] = pForwardInput[nInputSize * blockIdx.x + nIn];
	
	__syncthreads();
	
	unsigned int nUnifiedIn = blockDim.x * blockIdx.y + threadIdx.x;
	
	if (nUnifiedIn >= nInputSize)
		return;
	
	for (unsigned int nOut = threadIdx.x; nOut < nOutputSize; nOut += blockDim.x)
		pBiasDelta[nOutputSize * blockIdx.x + nOut] = pCopiedInput[nOut];
	
	float nValue = .0f;

	for (unsigned int nOut = 0; nOut < nOutputSize; ++nOut)
	{
		nValue += pCopiedInput[nOut] * pWeight[nInputSize * nOut + nUnifiedIn];
		pWeightDelta[nInputSize * nOutputSize * blockIdx.x + nInputSize * nOut + nUnifiedIn] = pCopiedInput[nOut] * pCopiedInput[nOutputSize + nUnifiedIn];
	}

	pBackwardOutput[nInputSize * blockIdx.x + nUnifiedIn] = nValue;
}



__global__ void kernel_LReLULayer_GPU_forward(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	const float vValue[2] =
	{
		.01f,
		1.f
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nUnifiedIndex] = vValue[pInput[nUnifiedIndex] > .0f] * pInput[nUnifiedIndex];
}

__global__ void kernel_LReLULayer_GPU_forwardBatch(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	const float vValue[2] =
	{
		.01f,
		1.f
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nSize * blockIdx.x + nUnifiedIndex] = vValue[pInput[nSize * blockIdx.x + nUnifiedIndex] > .0f] * pInput[nSize * blockIdx.x + nUnifiedIndex];
}

__global__ void kernel_LReLULayer_GPU_backwardBatch(unsigned int nSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	const float vValue[2] =
	{
		.01f,
		1.f
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pBackwardOutput[nSize * blockIdx.x + nUnifiedIndex] = pBackwardInput[nSize * blockIdx.x + nUnifiedIndex] * vValue[pForwardInput[nSize * blockIdx.x + nUnifiedIndex] > .0f];
}



__global__ void kernel_PReLULayer_GPU_forward(unsigned int nSize, const float *pInput, float *pOutput, float nParam)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	const float vValue[2] =
	{
		nParam,
		1.f
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nUnifiedIndex] = vValue[pInput[nUnifiedIndex] > .0f] * pInput[nUnifiedIndex];
}

__global__ void kernel_PReLULayer_GPU_forwardBatch(unsigned int nSize, const float *pInput, float *pOutput, float nParam)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	const float vValue[2] =
	{
		nParam,
		1.f
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nSize * blockIdx.x + nUnifiedIndex] = vValue[pInput[nSize * blockIdx.x + nUnifiedIndex] > .0f] * pInput[nSize * blockIdx.x + nUnifiedIndex];
}

__global__ void kernel_PReLULayer_GPU_backwardBatch(unsigned int nSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput, float nParam)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	const float vValue[2] =
	{
		nParam,
		1.f
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pBackwardOutput[nSize * blockIdx.x + nUnifiedIndex] = pBackwardInput[nSize * blockIdx.x + nUnifiedIndex] * vValue[pForwardInput[nSize * blockIdx.x + nUnifiedIndex] > .0f];
}



__global__ void kernel_ReLULayer_GPU_forward(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	const float vValue[2] =
	{
		.0f,
		1.f
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nUnifiedIndex] = vValue[pInput[nUnifiedIndex] > .0f] * pInput[nUnifiedIndex];
}

__global__ void kernel_ReLULayer_GPU_forwardBatch(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	const float vValue[2] =
	{
		.0f,
		1.f
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nSize * blockIdx.x + nUnifiedIndex] = vValue[pInput[nSize * blockIdx.x + nUnifiedIndex] > .0f] * pInput[nSize * blockIdx.x + nUnifiedIndex];
}

__global__ void kernel_ReLULayer_GPU_backwardBatch(unsigned int nSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	const float vValue[2] =
	{
		.0f,
		1.f
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pBackwardOutput[nSize * blockIdx.x + nUnifiedIndex] = pBackwardInput[nSize * blockIdx.x + nUnifiedIndex] * vValue[pForwardInput[nSize * blockIdx.x + nUnifiedIndex] > .0f];
}



__global__ void kernel_SigmoidLayer_GPU_forward(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nUnifiedIndex] = 1.f / (expf(-pInput[nUnifiedIndex]) + 1.f);
}

__global__ void kernel_SigmoidLayer_GPU_forwardBatch(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nSize * blockIdx.x + nUnifiedIndex] = 1.f / (expf(-pInput[nSize * blockIdx.x + nUnifiedIndex]) + 1.f);
}

__global__ void kernel_SigmoidLayer_GPU_backwardBatch(unsigned int nSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;

	float nValue = 1.f / (expf(-pForwardInput[nSize * blockIdx.x + nUnifiedIndex]) + 1.f);
	pBackwardOutput[nSize * blockIdx.x + nUnifiedIndex] = pBackwardInput[nSize * blockIdx.x + nUnifiedIndex] * nValue * (1.f - nValue);
}



__global__ void kernel_SwishLayer_GPU_forward(unsigned int nSize, const float *pInput, float *pOutput, float nBeta)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nUnifiedIndex] = pInput[nUnifiedIndex] / (expf(-nBeta * pInput[nUnifiedIndex]) + 1.f);
}

__global__ void kernel_SwishLayer_GPU_forwardBatch(unsigned int nSize, const float *pInput, float *pOutput, float nBeta)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nSize * blockIdx.x + nUnifiedIndex] = pInput[nSize * blockIdx.x + nUnifiedIndex] / (expf(-nBeta * pInput[nSize * blockIdx.x + nUnifiedIndex]) + 1.f);
}

__global__ void kernel_SwishLayer_GPU_backwardBatch(unsigned int nSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput, float nBeta)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;

	float nValue = 1.f / (expf(-nBeta * pForwardInput[nSize * blockIdx.x + nUnifiedIndex]) + 1.f);
	pBackwardOutput[nSize * blockIdx.x + nUnifiedIndex] = pBackwardInput[nSize * blockIdx.x + nUnifiedIndex] * (nValue + nBeta * pForwardInput[nSize * blockIdx.x + nUnifiedIndex] * nValue * (1.f - nValue));
}



__global__ void kernel_SoftmaxLayer_GPU_forward(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	float vValue[2] =
	{
		pInput[0],
		pInput[0]
	};

	for (unsigned int nIndex = 1; nIndex < nSize; ++nIndex)
	{
		vValue[1] = pInput[nIndex];
		vValue[0] = vValue[vValue[1] > vValue[0]];
	}

	vValue[1] = .0f;

	for (unsigned int nIndex = 0; nIndex < nSize; ++nIndex)
		vValue[1] += expf(pInput[nIndex] - vValue[0]);

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nUnifiedIndex] = expf(pInput[nUnifiedIndex] - vValue[0]) / vValue[1];
}

__global__ void kernel_SoftmaxLayer_GPU_forwardBatch(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	float vValue[2] =
	{
		pInput[0],
		pInput[0]
	};

	for (unsigned int nIndex = 1; nIndex < nSize; ++nIndex)
	{
		vValue[1] = pInput[nSize * blockIdx.x + nIndex];
		vValue[0] = vValue[vValue[1] > vValue[0]];
	}

	vValue[1] = .0f;

	for (unsigned int nIndex = 0; nIndex < nSize; ++nIndex)
		vValue[1] += expf(pInput[nSize * blockIdx.x + nIndex] - vValue[0]);

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nSize * blockIdx.x + nUnifiedIndex] = expf(pInput[nSize * blockIdx.x + nUnifiedIndex] - vValue[0]) / vValue[1];
}

__global__ void kernel_SoftmaxLayer_GPU_backwardBatch(unsigned int nSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput)
{
	extern __shared__ float vOutput[];
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};
	float vValue[2] =
	{
		pForwardInput[nSize * blockIdx.x],
		pForwardInput[nSize * blockIdx.x]
	};

	for (unsigned int nIndex = 1; nIndex < nSize; ++nIndex)
	{
		vValue[1] = pForwardInput[nSize * blockIdx.x + nIndex];
		vValue[0] = vValue[vValue[1] > vValue[0]];
	}

	vValue[1] = .0f;

	for (unsigned int nIndex = 0; nIndex < nSize; ++nIndex)
		vValue[1] += expf(pForwardInput[nSize * blockIdx.x + nIndex] - vValue[0]);

	for (unsigned int nIndex = threadIdx.x; nIndex < nSize; nIndex += blockDim.x)
		vOutput[nIndex] = expf(pForwardInput[nSize * blockIdx.x + nIndex] - vValue[0]) / vValue[1];

	__syncthreads();
	nUnifiedIndex = vMax[nUnifiedIndex < nSize];

	float nValue = .0f;

	for (unsigned int nIndex = 0; nIndex < nSize; ++nIndex)
	{
		vValue[0] = -vOutput[nIndex] * vOutput[nUnifiedIndex];
		vValue[1] = vOutput[nIndex] * (1.f - vOutput[nIndex]);
		nValue += pBackwardInput[nSize * blockIdx.x + nIndex] * vValue[nIndex == nUnifiedIndex];
	}

	pBackwardOutput[nSize * blockIdx.x + nUnifiedIndex] = nValue;
}



__global__ void kernel_TanhLayer_GPU_forward(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nUnifiedIndex] = tanhf(pInput[nUnifiedIndex]);
}

__global__ void kernel_TanhLayer_GPU_forwardBatch(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nSize * blockIdx.x + nUnifiedIndex] = tanhf(pInput[nSize * blockIdx.x + nUnifiedIndex]);
}

__global__ void kernel_TanhLayer_GPU_backwardBatch(unsigned int nSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];

	float nValue = tanhf(pForwardInput[nSize * blockIdx.x + nUnifiedIndex]);
	pBackwardOutput[nSize * blockIdx.x + nUnifiedIndex] = pBackwardInput[nSize * blockIdx.x + nUnifiedIndex] * (1.f - nValue * nValue);
}

#pragma endregion

void mergeBatch(std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_mergeBatch<<<sDimGrid, sDimBlock>>>(nBatchSize, nSize, sInput.address(), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("mergeBatch : %d\n", nError);
#endif
}

void updateParam(std::size_t nBiasSize, std::size_t nWeightSize, TinNet::GPUVector &sBias, TinNet::GPUVector &sWeight, const TinNet::GPUVector &sBiasDelta, const TinNet::GPUVector &sWeightDelta)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nBiasSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nBiasSize < 1024 ? nBiasSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_update<<<sDimGrid, sDimBlock>>>(nBiasSize, sBiasDelta.address(), sBias.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("updateParam__BIAS : %d\n", nError);
#endif

	sDimGrid.x = (nWeightSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nWeightSize < 1024 ? nWeightSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_update<<<sDimGrid, sDimBlock>>>(nWeightSize, sWeightDelta.address(), sWeight.address());

#if (_DEBUG)
	nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("updateParam__WEIGHT : %d\n", nError);
#endif
}

void updateParamFactor(std::size_t nBiasSize, std::size_t nWeightSize, TinNet::GPUVector &sBias, TinNet::GPUVector &sWeight, const TinNet::GPUVector &sBiasDelta, const TinNet::GPUVector &sWeightDelta, float nFactor)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nBiasSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nBiasSize < 1024 ? nBiasSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_updateFactor<<<sDimGrid, sDimBlock>>>(nBiasSize, sBiasDelta.address(), sBias.address(), nFactor);

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("kernel_updateFactor__BIAS : %d\n", nError);
#endif

	sDimGrid.x = (nWeightSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nWeightSize < 1024 ? nWeightSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_updateFactor<<<sDimGrid, sDimBlock>>>(nWeightSize, sWeightDelta.address(), sWeight.address(), nFactor);

#if (_DEBUG)
	nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("kernel_updateFactor__WEIGHT : %d\n", nError);
#endif
}

#pragma region Loss_GPU

void MSE_GPU_derivative(std::size_t nIndex, std::size_t nBatchSize, std::size_t nOutputSize, const TinNet::GPUVector &sOutput, const TinNet::GPUVector &sDesiredOutput, TinNet::GPUVector &sResult)
{
	std::size_t nSize = nBatchSize * nOutputSize;

	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_MSE_GPU_derivative<<<sDimGrid, sDimBlock>>>(nSize, nOutputSize, sOutput.address(), sDesiredOutput.address(nIndex), sResult.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("MSE_GPU_derivative : %d\n", nError);
#endif
}

void CE_GPU_derivative(std::size_t nIndex, std::size_t nBatchSize, std::size_t nOutputSize, const TinNet::GPUVector &sOutput, const TinNet::GPUVector &sDesiredOutput, TinNet::GPUVector &sResult)
{
	std::size_t nSize = nBatchSize * nOutputSize;

	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_CE_GPU_derivative<<<sDimGrid, sDimBlock>>>(nSize, nOutputSize, sOutput.address(), sDesiredOutput.address(nIndex), sResult.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("CE_GPU_derivative : %d\n", nError);
#endif
}

void MulticlassCE_GPU_derivative(std::size_t nIndex, std::size_t nBatchSize, std::size_t nOutputSize, const TinNet::GPUVector &sOutput, const TinNet::GPUVector &sDesiredOutput, TinNet::GPUVector &sResult)
{
	std::size_t nSize = nBatchSize * nOutputSize;

	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_MulticlassCE_GPU_derivative<<<sDimGrid, sDimBlock>>>(nSize, nOutputSize, sOutput.address(), sDesiredOutput.address(nIndex), sResult.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("MulticlassCE_GPU_derivative : %d\n", nError);
#endif
}

#pragma endregion

#pragma region Layer_GPU

void ConvLayer_GPU_forward(
	std::size_t nWidth,
	std::size_t nHeight,
	std::size_t nChannel,
	std::size_t nFilter,
	std::size_t nFilterWidth,
	std::size_t nFilterHeight,
	std::size_t nStrideHorizontal,
	std::size_t nStrideVertical,
	std::size_t nOutputWidth,
	std::size_t nOutputHeight,
	std::size_t nZeroPaddingHorizontalNegative,
	std::size_t nZeroPaddingVerticalNegative,
	const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput, const TinNet::GPUVector &sBias, const TinNet::GPUVector &sWeight)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nFilter;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nOutputWidth;
	sDimBlock.y = nOutputHeight;
	sDimBlock.z = 1;

	kernel_ConvLayer_GPU_forward<<<sDimGrid, sDimBlock>>>(
		nWidth,
		nHeight,
		nChannel,
		nFilterWidth,
		nFilterHeight,
		nStrideHorizontal,
		nStrideVertical,
		nZeroPaddingHorizontalNegative,
		nZeroPaddingVerticalNegative,
		sBias.address(), sWeight.address(), sInput.address(), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("ConvLayer_GPU_forward : %d\n", nError);
#endif
}

void ConvLayer_GPU_forwardBatch(
	std::size_t nIndex,
	std::size_t nBatchSize,
	std::size_t nWidth,
	std::size_t nHeight,
	std::size_t nChannel,
	std::size_t nFilter,
	std::size_t nFilterWidth,
	std::size_t nFilterHeight,
	std::size_t nStrideHorizontal,
	std::size_t nStrideVertical,
	std::size_t nOutputWidth,
	std::size_t nOutputHeight,
	std::size_t nZeroPaddingHorizontalNegative,
	std::size_t nZeroPaddingVerticalNegative,
	const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput, const TinNet::GPUVector &sBias, const TinNet::GPUVector &sWeight)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nFilter;
	sDimGrid.y = nBatchSize;
	sDimGrid.z = 1;

	sDimBlock.x = nOutputWidth;
	sDimBlock.y = nOutputHeight;
	sDimBlock.z = 1;

	kernel_ConvLayer_GPU_forwardBatch<<<sDimGrid, sDimBlock>>>(
		nWidth,
		nHeight,
		nChannel,
		nFilterWidth,
		nFilterHeight,
		nStrideHorizontal,
		nStrideVertical,
		nZeroPaddingHorizontalNegative,
		nZeroPaddingVerticalNegative,
		sBias.address(), sWeight.address(), sInput.address(nIndex), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("ConvLayer_GPU_forwardBatch : %d\n", nError);
#endif
}

TINNET_DLL void ConvLayer_GPU_backwardBatch(
	std::size_t nIndex,
	std::size_t nBatchSize,
	std::size_t nWidth,
	std::size_t nHeight,
	std::size_t nChannel,
	std::size_t nFilter,
	std::size_t nFilterWidth,
	std::size_t nFilterHeight,
	std::size_t nStrideHorizontal,
	std::size_t nStrideVertical,
	std::size_t nOutputWidth,
	std::size_t nOutputHeight,
	std::size_t nZeroPaddingHorizontalNegative,
	std::size_t nZeroPaddingVerticalNegative,
	const TinNet::GPUVector &sForwardInput, const TinNet::GPUVector &sBackwardInput, TinNet::GPUVector &sBackwardOutput, TinNet::GPUVector &sBiasDelta, TinNet::GPUVector &sWeightDelta, const TinNet::GPUVector &sWeight)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nFilter;
	sDimGrid.y = nBatchSize;
	sDimGrid.z = 1;

	sDimBlock.x = nOutputWidth;
	sDimBlock.y = nOutputHeight;
	sDimBlock.z = 1;

	kernel_ConvLayer_GPU_backwardBatch<<<sDimGrid, sDimBlock>>>(
		nWidth,
		nHeight,
		nChannel,
		nFilterWidth,
		nFilterHeight,
		nStrideHorizontal,
		nStrideVertical,
		nZeroPaddingHorizontalNegative,
		nZeroPaddingVerticalNegative,
		sForwardInput.address(nIndex), sBackwardInput.address(), sBackwardOutput.address(), sBiasDelta.address(), sWeightDelta.address(), sWeight.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("ConvLayer_GPU_backwardBatch : %d\n", nError);
#endif
}



void FullLayer_GPU_forward(std::size_t nInputSize, std::size_t nOutputSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput, const TinNet::GPUVector &sBias, const TinNet::GPUVector &sWeight)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nOutputSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nOutputSize < 1024 ? nOutputSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_FullLayer_GPU_forward<<<sDimGrid, sDimBlock, sizeof(float) * nInputSize>>>(nInputSize, nOutputSize, sBias.address(), sWeight.address(), sInput.address(), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("FullLayer_GPU_forward : %d\n", nError);
#endif
}

void FullLayer_GPU_forwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nInputSize, std::size_t nOutputSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput, const TinNet::GPUVector &sBias, const TinNet::GPUVector &sWeight)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nOutputSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nOutputSize < 1024 ? nOutputSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_FullLayer_GPU_forwardBatch<<<sDimGrid, sDimBlock, sizeof(float) * nInputSize>>>(nInputSize, nOutputSize, sBias.address(), sWeight.address(), sInput.address(nIndex), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("FullLayer_GPU_forwardBatch : %d\n", nError);
#endif
}

void FullLayer_GPU_backwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nInputSize, std::size_t nOutputSize, const TinNet::GPUVector &sForwardInput, const TinNet::GPUVector &sBackwardInput, TinNet::GPUVector &sBackwardOutput, TinNet::GPUVector &sBiasDelta, TinNet::GPUVector &sWeightDelta, const TinNet::GPUVector &sWeight)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nInputSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nInputSize < 1024 ? nInputSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_FullLayer_GPU_backwardBatch<<<sDimGrid, sDimBlock, sizeof(float) * (nInputSize + nOutputSize)>>>(nInputSize, nOutputSize, sForwardInput.address(nIndex), sBackwardInput.address(), sBackwardOutput.address(), sBiasDelta.address(), sWeightDelta.address(), sWeight.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("FullLayer_GPU_backwardBatch : %d\n", nError);
#endif
}



void LReLULayer_GPU_forward(std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_LReLULayer_GPU_forward<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("LReLULayer_GPU_forward : %d\n", nError);
#endif
}

void LReLULayer_GPU_forwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_LReLULayer_GPU_forwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(nIndex), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("LReLULayer_GPU_forwardBatch : %d\n", nError);
#endif
}

void LReLULayer_GPU_backwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sForwardInput, const TinNet::GPUVector &sBackwardInput, TinNet::GPUVector &sBackwardOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_LReLULayer_GPU_backwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sForwardInput.address(nIndex), sBackwardInput.address(), sBackwardOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("LReLULayer_GPU_backwardBatch : %d\n", nError);
#endif
}



void PReLULayer_GPU_forward(std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput, float nParam)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_PReLULayer_GPU_forward<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(), sOutput.address(), nParam);

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("PReLULayer_GPU_forward : %d\n", nError);
#endif
}

void PReLULayer_GPU_forwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput, float nParam)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_PReLULayer_GPU_forwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(nIndex), sOutput.address(), nParam);

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("PReLULayer_GPU_forwardBatch : %d\n", nError);
#endif
}

void PReLULayer_GPU_backwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sForwardInput, const TinNet::GPUVector &sBackwardInput, TinNet::GPUVector &sBackwardOutput, float nParam)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_PReLULayer_GPU_backwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sForwardInput.address(nIndex), sBackwardInput.address(), sBackwardOutput.address(), nParam);

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("PReLULayer_GPU_backwardBatch : %d\n", nError);
#endif
}



void ReLULayer_GPU_forward(std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_ReLULayer_GPU_forward<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("ReLULayer_GPU_forward : %d\n", nError);
#endif
}

void ReLULayer_GPU_forwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_ReLULayer_GPU_forwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(nIndex), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("ReLULayer_GPU_forwardBatch : %d\n", nError);
#endif
}

void ReLULayer_GPU_backwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sForwardInput, const TinNet::GPUVector &sBackwardInput, TinNet::GPUVector &sBackwardOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_ReLULayer_GPU_backwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sForwardInput.address(nIndex), sBackwardInput.address(), sBackwardOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("ReLULayer_GPU_backwardBatch : %d\n", nError);
#endif
}



void SigmoidLayer_GPU_forward(std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SigmoidLayer_GPU_forward<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SigmoidLayer_GPU_forward : %d\n", nError);
#endif
}

void SigmoidLayer_GPU_forwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SigmoidLayer_GPU_forwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(nIndex), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SigmoidLayer_GPU_forwardBatch : %d\n", nError);
#endif
}

void SigmoidLayer_GPU_backwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sForwardInput, const TinNet::GPUVector &sBackwardInput, TinNet::GPUVector &sBackwardOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SigmoidLayer_GPU_backwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sForwardInput.address(nIndex), sBackwardInput.address(), sBackwardOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SigmoidLayer_GPU_backwardBatch : %d\n", nError);
#endif
}



void SwishLayer_GPU_forward(std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput, float nBeta)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SwishLayer_GPU_forward<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(), sOutput.address(), nBeta);

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SwishLayer_GPU_forward : %d\n", nError);
#endif
}

void SwishLayer_GPU_forwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput, float nBeta)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SwishLayer_GPU_forwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(nIndex), sOutput.address(), nBeta);

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SwishLayer_GPU_forwardBatch : %d\n", nError);
#endif
}

void SwishLayer_GPU_backwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sForwardInput, const TinNet::GPUVector &sBackwardInput, TinNet::GPUVector &sBackwardOutput, float nBeta)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SwishLayer_GPU_backwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sForwardInput.address(nIndex), sBackwardInput.address(), sBackwardOutput.address(), nBeta);

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SwishLayer_GPU_backwardBatch : %d\n", nError);
#endif
}



void SoftmaxLayer_GPU_forward(std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SoftmaxLayer_GPU_forward<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SoftmaxLayer_GPU_forward : %d\n", nError);
#endif
}

void SoftmaxLayer_GPU_forwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SoftmaxLayer_GPU_forwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(nIndex), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SoftmaxLayer_GPU_forwardBatch : %d\n", nError);
#endif
}

void SoftmaxLayer_GPU_backwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sForwardInput, const TinNet::GPUVector &sBackwardInput, TinNet::GPUVector &sBackwardOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SoftmaxLayer_GPU_backwardBatch<<<sDimGrid, sDimBlock, sizeof(float) * nSize>>>(nSize, sForwardInput.address(nIndex), sBackwardInput.address(), sBackwardOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SoftmaxLayer_GPU_backwardBatch : %d\n", nError);
#endif
}



void TanhLayer_GPU_forward(std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_TanhLayer_GPU_forward<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("TanhLayer_GPU_forward : %d\n", nError);
#endif
}

void TanhLayer_GPU_forwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sInput, TinNet::GPUVector &sOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_TanhLayer_GPU_forwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sInput.address(nIndex), sOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("TanhLayer_GPU_forwardBatch : %d\n", nError);
#endif
}

void TanhLayer_GPU_backwardBatch(std::size_t nIndex, std::size_t nBatchSize, std::size_t nSize, const TinNet::GPUVector &sForwardInput, const TinNet::GPUVector &sBackwardInput, TinNet::GPUVector &sBackwardOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_TanhLayer_GPU_backwardBatch<<<sDimGrid, sDimBlock>>>(nSize, sForwardInput.address(nIndex), sBackwardInput.address(), sBackwardOutput.address());

#if (_DEBUG)
	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("TanhLayer_GPU_backwardBatch : %d\n", nError);
#endif
}

#pragma endregion