#include "hip/hip_runtime.h"

/*
	2018.01.20
	Created by AcrylicShrimp.
*/

#include "TinNet_GPU_Backend.h"

#include "cuda_device_runtime_api.h"
#include "hip/hip_runtime.h"
#include "device_atomic_functions.h"
#include "hip/device_functions.h"
#include ""
#include "math_functions.h"

#include <cstdio>

#pragma region Kernel

__global__ void kernel_mergeBatch(unsigned int nBatchSize, unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;

	for (unsigned int nBatch = 1; nBatch < nBatchSize; ++nBatch)
		pOutput[nUnifiedIndex] += pInput[nSize * nBatch + nUnifiedIndex];
}



__global__ void kernel_MSE_GPU_derivative(unsigned int nSize, unsigned int nOutputSize, const float *pOutput, const float *pDesiredOutput, float *pResult)
{
	unsigned int nUnifiedOut = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedOut
	};

	nUnifiedOut = vMax[nUnifiedOut < nSize];
	pResult[nUnifiedOut] = 2.f / nOutputSize * (pOutput[nUnifiedOut] - pDesiredOutput[nUnifiedOut]);
}



__global__ void kernel_FullLayer_GPU_forward(unsigned int nInputSize, unsigned int nOutputSize, const float *pBias, const float *pWeight, const float *pInput, float *pOutput)
{
	extern __shared__ float pCopiedInput[];

	for (unsigned int nIn = threadIdx.x; nIn < nInputSize; nIn += blockDim.x)
		pCopiedInput[nIn] = pInput[nIn];

	__syncthreads();

	unsigned int nUnifiedOut = blockDim.x * blockIdx.x + threadIdx.x;

	if (nUnifiedOut >= nOutputSize)
		return;

	float nValue = pBias[nUnifiedOut];

	unsigned int nWeightIndex = nUnifiedOut * nInputSize;

	for (unsigned int nIn = 0; nIn < nInputSize; ++nIn)
		nValue += pCopiedInput[nIn] * pWeight[nWeightIndex + nIn];

	pOutput[nUnifiedOut] = nValue;
}

__global__ void kernel_FullLayer_GPU_forwardBatch(unsigned int nInputSize, unsigned int nOutputSize, const float *pBias, const float *pWeight, const float *pInput, float *pOutput)
{
	extern __shared__ float pCopiedInput[];

	for (unsigned int nIn = threadIdx.x; nIn < nInputSize; nIn += blockDim.x)
		pCopiedInput[nIn] = pInput[nInputSize * blockIdx.x + nIn];

	__syncthreads();
	
	unsigned int nUnifiedOut = blockDim.x * blockIdx.y + threadIdx.x;
	
	if (nUnifiedOut >= nOutputSize)
		return;
	
	float nValue = pBias[nUnifiedOut];
	
	for (unsigned int nIn = 0; nIn < nInputSize; ++nIn)
		nValue += pCopiedInput[nIn] * pWeight[nUnifiedOut * nInputSize + nIn];
	
	pOutput[nOutputSize * blockIdx.x + nUnifiedOut] = nValue;
}

__global__ void kernel_FullLayer_GPU_backwardBatch(unsigned int nInputSize, unsigned int nOutputSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput, float *pBiasDelta, float *pWeightDelta, const float *pWeight)
{
	extern __shared__ float pCopiedInput[];

	for (unsigned int nOut = threadIdx.x; nOut < nOutputSize; nOut += blockDim.x)
		pCopiedInput[nOut] = pBackwardInput[nOutputSize * blockIdx.x + nOut];

	for (unsigned int nIn = threadIdx.x; nIn < nInputSize; nIn += blockDim.x)
		pCopiedInput[nOutputSize + nIn] = pForwardInput[nInputSize * blockIdx.x + nIn];

	__syncthreads();
	
	unsigned int nUnifiedIn = blockDim.x * blockIdx.y + threadIdx.x;
	
	if (nUnifiedIn >= nInputSize)
		return;
	
	for (unsigned int nOut = threadIdx.x; nOut < nOutputSize; nOut += blockDim.x)
		pBiasDelta[nOutputSize * blockIdx.x + nOut] = pCopiedInput[nOut];
	
	float nValue = .0f;

	for (unsigned int nOut = 0; nOut < nOutputSize; ++nOut)
	{
		nValue = pCopiedInput[nOut] * pWeight[nInputSize * nOut + nUnifiedIn];
		pWeightDelta[nInputSize * nOutputSize * blockIdx.x + nInputSize * nOut + nUnifiedIn] = pCopiedInput[nOut] * pCopiedInput[nOutputSize + nUnifiedIn];
	}

	pBackwardOutput[nInputSize * blockIdx.x + nUnifiedIn] = nValue;
}

__global__ void kernel_FullLayer_GPU_update(unsigned int nParamSize, const float *pParamDelta, float *pParam)
{
	unsigned int nUnifiedOut = blockDim.x * blockIdx.x + threadIdx.x;

	if (nUnifiedOut >= nParamSize)
		return;

	pParam[nUnifiedOut] += pParamDelta[nUnifiedOut];
}

__global__ void kernel_FullLayer_GPU_updateFactor(unsigned int nParamSize, const float *pParamDelta, float *pParam, float nFactor)
{
	unsigned int nUnifiedOut = blockDim.x * blockIdx.x + threadIdx.x;

	if (nUnifiedOut >= nParamSize)
		return;

	pParam[nUnifiedOut] += nFactor * pParamDelta[nUnifiedOut];
}



__global__ void kernel_SigmoidLayer_GPU_forward(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nUnifiedIndex] = 1.f / (expf(-pInput[nUnifiedIndex]) + 1.f);
}

__global__ void kernel_SigmoidLayer_GPU_forwardBatch(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nSize * blockIdx.x + nUnifiedIndex] = 1.f / (expf(-pInput[nSize * blockIdx.x + nUnifiedIndex]) + 1.f);
}

__global__ void kernel_SigmoidLayer_GPU_backwardBatch(unsigned int nSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;

	float nValue = 1.f / (expf(-pForwardInput[nSize * blockIdx.x + nUnifiedIndex]) + 1.f);
	pBackwardOutput[nSize * blockIdx.x + nUnifiedIndex] = pBackwardInput[nSize * blockIdx.x + nUnifiedIndex] * nValue * (1.f - nValue);
}



__global__ void kernel_TanhLayer_GPU_forward(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nUnifiedIndex] = tanhf(pInput[nUnifiedIndex]);
}

__global__ void kernel_TanhLayer_GPU_forwardBatch(unsigned int nSize, const float *pInput, float *pOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;
	unsigned int vMax[2] =
	{
		nSize - 1,
		nUnifiedIndex
	};

	nUnifiedIndex = vMax[nUnifiedIndex < nSize];
	pOutput[nSize * blockIdx.x + nUnifiedIndex] = tanhf(pInput[nSize * blockIdx.x + nUnifiedIndex]);
}

__global__ void kernel_TanhLayer_GPU_backwardBatch(unsigned int nSize, const float *pForwardInput, const float *pBackwardInput, float *pBackwardOutput)
{
	unsigned int nUnifiedIndex = blockDim.x * blockIdx.y + threadIdx.x;

	float nValue = tanhf(pForwardInput[nSize * blockIdx.x + nUnifiedIndex]);
	pBackwardOutput[nSize * blockIdx.x + nUnifiedIndex] = pBackwardInput[nSize * blockIdx.x + nUnifiedIndex] * (1.f - nValue * nValue);
}

#pragma endregion

void mergeBatch(std::size_t nBatchSize, std::size_t nSize, hipDeviceptr_t pInput, hipDeviceptr_t pOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_mergeBatch<<<sDimGrid, sDimBlock>>>(nBatchSize, nSize, (const float *)pInput, (float *)pOutput);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("mergeBatch : %d\n", nError);
}

#pragma region MSE_GPU

void MSE_GPU_derivative(std::size_t nBatchSize, std::size_t nOutputSize, hipDeviceptr_t pOutput, hipDeviceptr_t pDesiredOutput, hipDeviceptr_t pResult)
{
	std::size_t nSize = nBatchSize * nOutputSize;

	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_MSE_GPU_derivative<<<sDimGrid, sDimBlock>>>(nSize, nOutputSize, (const float *)pOutput, (const float *)pDesiredOutput, (float *)pResult);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("MSE_GPU_derivative : %d\n", nError);
}

#pragma endregion

#pragma region Layer_GPU

void FullLayer_GPU_forward(std::size_t nInputSize, std::size_t nOutputSize, hipDeviceptr_t pInput, hipDeviceptr_t pOutput, hipDeviceptr_t pBias, hipDeviceptr_t pWeight)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nOutputSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nOutputSize < 1024 ? nOutputSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_FullLayer_GPU_forward<<<sDimGrid, sDimBlock, sizeof(float) * nInputSize>>>(nInputSize, nOutputSize, (const float *)pBias, (const float *)pWeight, (const float *)pInput, (float *)pOutput);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("FullLayer_GPU_forward : %d\n", nError);
}

void FullLayer_GPU_forwardBatch(std::size_t nBatchSize, std::size_t nInputSize, std::size_t nOutputSize, hipDeviceptr_t pInput, hipDeviceptr_t pOutput, hipDeviceptr_t pBias, hipDeviceptr_t pWeight)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nOutputSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nOutputSize < 1024 ? nOutputSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_FullLayer_GPU_forwardBatch<<<sDimGrid, sDimBlock, sizeof(float) * nInputSize>>>(nInputSize, nOutputSize, (const float *)pBias, (const float *)pWeight, (const float *)pInput, (float *)pOutput);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("FullLayer_GPU_forwardBatch : %d\n", nError);
}

void FullLayer_GPU_backwardBatch(std::size_t nBatchSize, std::size_t nInputSize, std::size_t nOutputSize, hipDeviceptr_t pForwardInput, hipDeviceptr_t pBackwardInput, hipDeviceptr_t pBackwardOutput, hipDeviceptr_t pBiasDelta, hipDeviceptr_t pWeightDelta, hipDeviceptr_t pWeight)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nInputSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nInputSize < 1024 ? nInputSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_FullLayer_GPU_backwardBatch<<<sDimGrid, sDimBlock, sizeof(float) * (nInputSize + nOutputSize)>>>(nInputSize, nOutputSize, (const float *)pForwardInput, (const float *)pBackwardInput, (float *)pBackwardOutput, (float *)pBiasDelta, (float *)pWeightDelta, (const float *)pWeight);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("FullLayer_GPU_backwardBatch : %d\n", nError);
}

void FullLayer_GPU_updateParam(std::size_t nBiasSize, std::size_t nWeightSize, hipDeviceptr_t pBias, hipDeviceptr_t pWeight, hipDeviceptr_t pBiasDelta, hipDeviceptr_t pWeightDelta)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nBiasSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nBiasSize < 1024 ? nBiasSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_FullLayer_GPU_update << <sDimGrid, sDimBlock >> > (nBiasSize, (const float *)pBiasDelta, (float *)pBias);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("FullLayer_GPU_updateParam__BIAS : %d\n", nError);

	sDimGrid.x = (nWeightSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nWeightSize < 1024 ? nWeightSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_FullLayer_GPU_update<<<sDimGrid, sDimBlock>>>(nWeightSize, (const float *)pWeightDelta, (float *)pWeight);

	nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("FullLayer_GPU_updateParam__WEIGHT : %d\n", nError);
}

void FullLayer_GPU_updateParamFactor(std::size_t nBiasSize, std::size_t nWeightSize, hipDeviceptr_t pBias, hipDeviceptr_t pWeight, hipDeviceptr_t pBiasDelta, hipDeviceptr_t pWeightDelta, float nFactor)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nBiasSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nBiasSize < 1024 ? nBiasSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_FullLayer_GPU_updateFactor<<<sDimGrid, sDimBlock>>>(nBiasSize, (const float *)pBiasDelta, (float *)pBias, nFactor);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("FullLayer_GPU_updateParamFactor__BIAS : %d\n", nError);

	sDimGrid.x = (nWeightSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nWeightSize < 1024 ? nWeightSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_FullLayer_GPU_updateFactor<<<sDimGrid, sDimBlock>>>(nWeightSize, (const float *)pWeightDelta, (float *)pWeight, nFactor);

	nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("FullLayer_GPU_updateParamFactor__WEIGHT : %d\n", nError);
}



void SigmoidLayer_GPU_forward(std::size_t nSize, hipDeviceptr_t pInput, hipDeviceptr_t pOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SigmoidLayer_GPU_forward<<<sDimGrid, sDimBlock>>>(nSize, (const float *)pInput, (float *)pOutput);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SigmoidLayer_GPU_forward : %d\n", nError);
}

void SigmoidLayer_GPU_forwardBatch(std::size_t nBatchSize, std::size_t nSize, hipDeviceptr_t pInput, hipDeviceptr_t pOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SigmoidLayer_GPU_forwardBatch<<<sDimGrid, sDimBlock>>>(nSize, (const float *)pInput, (float *)pOutput);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SigmoidLayer_GPU_forwardBatch : %d\n", nError);
}

void SigmoidLayer_GPU_backwardBatch(std::size_t nBatchSize, std::size_t nSize, hipDeviceptr_t pForwardInput, hipDeviceptr_t pBackwardInput, hipDeviceptr_t pBackwardOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_SigmoidLayer_GPU_backwardBatch<<<sDimGrid, sDimBlock>>>(nSize, (const float *)pForwardInput, (const float *)pBackwardInput, (float *)pBackwardOutput);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("SigmoidLayer_GPU_backwardBatch : %d\n", nError);
}



void TanhLayer_GPU_forward(std::size_t nSize, hipDeviceptr_t pInput, hipDeviceptr_t pOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = (nSize - 1) / 1024 + 1;
	sDimGrid.y = 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_TanhLayer_GPU_forward<<<sDimGrid, sDimBlock>>>(nSize, (const float *)pInput, (float *)pOutput);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("TanhLayer_GPU_forward : %d\n", nError);
}

void TanhLayer_GPU_forwardBatch(std::size_t nBatchSize, std::size_t nSize, hipDeviceptr_t pInput, hipDeviceptr_t pOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_TanhLayer_GPU_forwardBatch<<<sDimGrid, sDimBlock>>>(nSize, (const float *)pInput, (float *)pOutput);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("TanhLayer_GPU_forwardBatch : %d\n", nError);
}

void TanhLayer_GPU_backwardBatch(std::size_t nBatchSize, std::size_t nSize, hipDeviceptr_t pForwardInput, hipDeviceptr_t pBackwardInput, hipDeviceptr_t pBackwardOutput)
{
	uint3 sDimGrid;
	uint3 sDimBlock;

	sDimGrid.x = nBatchSize;
	sDimGrid.y = (nSize - 1) / 1024 + 1;
	sDimGrid.z = 1;

	sDimBlock.x = nSize < 1024 ? nSize : 1024;
	sDimBlock.y = 1;
	sDimBlock.z = 1;

	kernel_TanhLayer_GPU_backwardBatch<<<sDimGrid, sDimBlock>>>(nSize, (const float *)pForwardInput, (const float *)pBackwardInput, (float *)pBackwardOutput);

	hipError_t nError = hipGetLastError();

	if (nError != hipError_t::hipSuccess)
		printf("TanhLayer_GPU_backwardBatch : %d\n", nError);
}

#pragma endregion